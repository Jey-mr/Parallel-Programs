
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>
using namespace std;

/*
__device__ double atomicAddUD(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

     } while (assumed != old);

    return __longlong_as_double(old);
}
*/

__global__ void mykernal(double *dA1, double *dx, double *dx1, int n){
	int i, j, k;
	
	i = threadIdx.x;
	for(j=0,k=0; k<n; j++,k++){
		if(j==i && j==n-1)
			break;
		else if(j == i)
			k++;
				
		dx1[i] += (*(dA1 + i*n + j) * dx[k]);
		//dx1[i] += (dA1[i][j] * dx[k]);
	}
		
	dx1[i] += *(dA1 + i*n + n-1);	
	//dx1[i] += dA1[i][n-1];
}

__global__ void mykernal2(double *dx, double *dx1, double *dxp){
	int i = threadIdx.x;
	
	dxp[i] = dx[i];
	dx[i] = dx1[i];
	dx1[i] = 0;
}


int main(){
	struct timeval start, end;
	gettimeofday(&start, NULL);
	
	ifstream fin("input.txt");
	
	int i, j, k, n, l, limit;
	double difference;
	fin >> n;
	
	double *A = (double*)malloc(sizeof(double)*n*n); 
	double *A_dash = (double*)malloc(sizeof(double)*n*n);
	double *b = (double*)malloc(sizeof(double)*n); 
	double *x = (double*)malloc(sizeof(double)*n); 
	double *x_dash = (double*)malloc(sizeof(double)*n); 
	
		
	for(i=0; i<n; i++)
		for(j=0; j<n; j++)
			fin >> *(A + i*n + j);
			
	for(i=0; i<n; i++){
		fin >> b[i];
		x[i] = x_dash[i] = 0;	
	}
		
/*	cout << "Matrix_A:" << endl;
	for(i=0; i<n; i++){
		for(j=0; j<n; j++)
			cout << *(A + i*n + j) << " ";
		
		cout << endl;
	}
	cout << endl;
	
	cout << "Vector_b:" << endl;
	for(i=0; i<n; i++)
		cout << b[i] << " ";
	cout << endl << endl;	*/
	
	for(i=0; i<n; i++){
		for(j=0,k=0; k<n; j++,k++){
			if(j==i && j==n-1)
				break;
			else if(j == i)
				k++;
			
			*(A_dash + i*n + j) = -1 * *(A + i*n + k) / *(A + i*n + i);				
		}
		
		*(A_dash + i*n + j) = b[i] / *(A + i*n + i);
	}
	
/*	cout << "Matrix_A_dash:" << endl;
	for(i=0; i<n; i++){
		for(j=0; j<n; j++)
			cout << A_dash[i][j] << " ";
		
		cout << endl;
	}
	cout << endl;	*/
	
	double *dA1, *dx, *dx1, *dxp;
	
	hipMalloc((void**)&dA1, sizeof(double)*n*n);
	hipMalloc((void**)&dx, sizeof(double)*n);
	hipMalloc((void**)&dx1, sizeof(double)*n);
	hipMalloc((void**)&dxp, sizeof(double)*n);
	
	hipMemcpy(dA1, A_dash, sizeof(double)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(dx, x, sizeof(double)*n, hipMemcpyHostToDevice);
	hipMemcpy(dx1, x_dash, sizeof(double)*n, hipMemcpyHostToDevice);
	hipMemcpy(dxp, x_dash, sizeof(double)*n, hipMemcpyHostToDevice);
	
	limit = 5*n*n;
	for(l=0; l<limit; l++){
		mykernal<<<1, n>>>(dA1, dx, dx1, n);
		hipDeviceSynchronize();
		
		mykernal2<<<1, n>>>(dx, dx1, dxp);
		hipDeviceSynchronize();	
	}
	
	hipMemcpy(x, dx, sizeof(double)*n, hipMemcpyDeviceToHost);
	hipMemcpy(x_dash, dxp, sizeof(double)*n, hipMemcpyDeviceToHost);
	hipFree(dA1);
	hipFree(dx);
	hipFree(dx1);	
	hipFree(dxp);
	
	difference = 0;
	for(i=0; i<n; i++)
		difference += abs(abs(x[i]) - abs(x_dash[i]));
	
	//cout << "Difference: " << difference << endl;
	
/*	if(difference/(5*n*n) > 1)
		cout << "it will not converge" << endl;	
	else{
		cout << "Vector_x:" << endl;
		
		for(i=0; i<n; i++)
			cout << x[i] << " ";
		cout << endl << endl;
	}	*/
	
	gettimeofday(&end, NULL);
	long seconds = (end.tv_sec - start.tv_sec);
	long micros = ((seconds * 1000000) + end.tv_usec) - start.tv_usec;
	double time = (double)micros / 1000000; 
	
	cout << "Cuda: " << time << endl;
	
	return 0;
}
