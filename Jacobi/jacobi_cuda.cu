
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
using namespace std;

__device__ double atomicAddUD(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

     } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void mykernal(double *dA1, double *dx, double *dx1, int n){
	int i, j;
	double value;
	i = blockIdx.x;
	j = threadIdx.x;
	
	if(j < i)
		value = *(dA1 + i*n + j) * dx[j];
	else if(j < n-1)
		value = *(dA1 + i*n + j) * dx[j+1];
	else 
		value = *(dA1 + i*n + j);
		
	atomicAddUD(&dx1[i], value);
}

__global__ void mykernal2(double *dx, double *dx1, double *d){
	int i = threadIdx.x;
	double diff = abs(abs(dx[i]) - abs(dx1[i]));
	
	atomicAddUD(d, diff);
	
	dx[i] = dx1[i];
	dx1[i] = 0;
}


int main(){
	ifstream fin("input.txt");
	
	int i, j, k, n, l, limit;
	double difference;
	fin >> n;
	
	double *A = (double*)malloc(sizeof(double)*n*n); 
	double *A_dash = (double*)malloc(sizeof(double)*n*n);
	double *b = (double*)malloc(sizeof(double)*n); 
	double *x = (double*)malloc(sizeof(double)*n); 
	double *x_dash = (double*)malloc(sizeof(double)*n); 
	
		
	for(i=0; i<n; i++)
		for(j=0; j<n; j++)
			fin >> *(A + i*n + j);
			
	for(i=0; i<n; i++){
		fin >> b[i];
		x[i] = x_dash[i] = 0;	
	}
		
	cout << "Matrix_A:" << endl;
	for(i=0; i<n; i++){
		for(j=0; j<n; j++)
			cout << *(A + i*n + j) << " ";
		
		cout << endl;
	}
	cout << endl;
	
	cout << "Vector_b:" << endl;
	for(i=0; i<n; i++)
		cout << b[i] << " ";
	cout << endl << endl;	
	
	for(i=0; i<n; i++){
		for(j=0,k=0; k<n; j++,k++){
			if(j==i && j==n-1)
				break;
			else if(j == i)
				k++;
			
			*(A_dash + i*n + j) = -1 * *(A + i*n + k) / *(A + i*n + i);				
		}
		
		*(A_dash + i*n + j) = b[i] / *(A + i*n + i);
	}
	
/*	cout << "Matrix_A_dash:" << endl;
	for(i=0; i<n; i++){
		for(j=0; j<n; j++)
			cout << A_dash[i][j] << " ";
		
		cout << endl;
	}
	cout << endl;	*/
	
	double *dA1, *dx, *dx1, *d;
	
	hipMalloc((void**)&dA1, sizeof(double)*n*n);
	hipMalloc((void**)&dx, sizeof(double)*n);
	hipMalloc((void**)&dx1, sizeof(double)*n);
	hipMalloc((void**)&d, sizeof(double));
	
	hipMemcpy(dA1, A_dash, sizeof(double)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(dx, x, sizeof(double)*n, hipMemcpyHostToDevice);
	hipMemcpy(dx1, x_dash, sizeof(double)*n, hipMemcpyHostToDevice);
	
	limit = 5*n*n;
	for(l=0; l<limit; l++){
		mykernal<<<n, n>>>(dA1, dx, dx1, n);
		hipDeviceSynchronize();
		
		difference = 0;
		hipMemcpy(d, &difference, sizeof(double), hipMemcpyHostToDevice);
		mykernal2<<<1, n>>>(dx, dx1, d);
		hipDeviceSynchronize();		
	}
	
	hipMemcpy(x, dx, sizeof(double)*n, hipMemcpyDeviceToHost);
	hipMemcpy(&difference, d, sizeof(double), hipMemcpyDeviceToHost);
	hipFree(dA1);
	hipFree(dx);
	hipFree(dx1);	
	hipFree(d);
	
	//cout << "Difference: " << difference << endl;
	
	if(difference > 1)
		cout << "it will not converge" << endl;	
	else{
		cout << "Vector_x:" << endl;
		
		for(i=0; i<n; i++)
			cout << x[i] << " ";
		cout << endl << endl;
	}
	
	return 0;
}
